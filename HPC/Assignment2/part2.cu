#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>
using namespace std;


int random_in_range( int minimum, int maximum )
{
  thread_local std::ranlux48 rng( 
    std::chrono::system_clock::now().time_since_epoch().count() );
  return std::uniform_int_distribution <int> ( minimum, maximum )( rng );
}

__global__
void matrixVector(int *vec, int *mat, int *result, int n, int m)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int sum=0;
    
    if(tid <= n) {
        for(int i=0; i<n; i++) {
            sum += vec[i]*mat[(i*m) + tid];
        }
        result[tid] = sum;
    }
}

void maxtrixVector_cpu(int *vec, int *mat, int *result, int n, int m) {
    for(int i = 0 ; i < n ; i++) {
        long sum = 0;
        for(int j = 0 ; j < m ; j++) {
            sum = sum + mat[j*m+i] * vec[j];  
        }
        result[i] = sum;
    }
}

void init_array(int *a, int n) {
    for(int i=0; i<n; i++)
      a[i] = random_in_range(10,40);
}

void init_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<m; j++) {
            a[i*m + j] = random_in_range(10, 40);        
        }
    }
}

void print_array(int *a, int n) {
    for(int i=0; i<n; i++) {
        cout<<a[i]<<" ";
    }
    cout<<endl;
}

void print_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<m; j++)
          cout<<"  "<<a[i*m + j];
        cout<<endl;
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    
    int n = 100;
    int m = 100;
    
    a = new int[n];
    b = new int[n*m];
    c = new int[m];
    
    init_array(a, n);
    init_matrix(b, n, m);
        
    cout<<"Initial vector array : "<<endl;
    print_array(a, n);
    cout<<endl;
    cout<<"Initial matrix : "<<endl;
    print_matrix(b, n, m);
    cout<<endl;
    
    hipMalloc(&a_dev, sizeof(int)*n);
    hipMalloc(&b_dev, sizeof(int)*n*m);
    hipMalloc(&c_dev, sizeof(int)*m);
    
    hipMemcpy(a_dev, a, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, sizeof(int)*n*m, hipMemcpyHostToDevice);
    
    float gpu_elapsed_time;
    hipEvent_t gpu_start,gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);
    matrixVector<<<m, 1>>>(a_dev, b_dev, c_dev, n, m);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    cout<<"GPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;

    hipMemcpy(c, c_dev, sizeof(int)*m, hipMemcpyDeviceToHost);
    
    cout<<"GPU Resultant vector : ";
    print_array(c, m);
    cout<<endl;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);
    maxtrixVector_cpu(a, b, c, n, m);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    cout<<"CPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;
    
    cout<<"CPU Resultant vector : ";
    for(int i = 0 ; i < n ; i++) {
        cout<<c[i]<<" ";
    }
    cout<<endl;
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    
    delete[] a;
    delete[] b;
    delete[] c;
    
    return 0;
}
