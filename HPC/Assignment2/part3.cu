#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <random>
using namespace std;


int random_in_range( int minimum, int maximum )
{
  thread_local std::ranlux48 rng( 
    std::chrono::system_clock::now().time_since_epoch().count() );
  return std::uniform_int_distribution <int> ( minimum, maximum )( rng );
}

__global__
void matrixMultiplication(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int sum=0;
   
    if(col<k && row<m) {
      for(int j=0;j<n;j++)
      {
          sum += a[row*n+j] * b[j*k+col];
      }
      c[k*row+col]=sum;
    }
    
}

void matrix_multiplication_cpu(int *a, int *b, int *c, int m, int n, int k) {
    for(int i = 0 ; i < m ; i++) {
        for(int j = 0 ; j < n ; j++) {
            long result = 0;
            for(int p = 0 ; p < k ; p++) {
                result=result+a[i*k+p]*b[p*k+j]; 
            }
            c[k*i+j] = result;
        }
    }
}

void init_result(int *a, int m, int k) {
    for(int i=0; i<m; i++) {
      for(int j=0; j<k; j++) {
        a[i*k + j] = 0;
      }
    }
}

void init_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
      for(int j=0; j<m; j++) {
        a[i*m + j] = random_in_range(10,30);
      }
    }
}

void print_matrix(int *a, int n, int m) {
    for(int i=0; i<n; i++) {
      for(int j=0; j<m; j++) {
        cout<<"  "<<a[i*m + j];
      }
      cout<<endl;
    }
    cout<<endl;
}

int main()
{
    
    int *a,*b,*c;
    int *a_dev,*b_dev,*c_dev;
    int m=30, n=30, k=30;
    
    a = new int[m*n];
    b = new int[n*k];
    c = new int[m*k];
    
    init_matrix(a, m, n);
    init_matrix(b, n ,k);
    init_result(c, m, k);
    
    cout<<"First matrix : "<<endl;
    print_matrix(a, m, n);
    cout<<"Second matrix : "<<endl;
    print_matrix(b, n, k);
    
    hipMalloc(&a_dev, sizeof(int)*m*n);
    hipMalloc(&b_dev, sizeof(int)*n*k);
    hipMalloc(&c_dev, sizeof(int)*m*k);
       
    hipMemcpy(a_dev, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, sizeof(int)*n*k, hipMemcpyHostToDevice);
    
    dim3 dimGrid(1,1);
    dim3 dimBlock(n,n);
    
    float gpu_elapsed_time;
    hipEvent_t gpu_start,gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);
    matrixMultiplication<<<dimGrid, dimBlock>>>(a_dev,b_dev,c_dev, m, n, k);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    cout<<"GPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;
    
    hipMemcpy(c, c_dev, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    
    cout<<"GPU Result : "<<endl;
    print_matrix(c, m, k);
    cout<<endl;

    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start,0);
    matrix_multiplication_cpu(a, b, c, m, n, k);
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);
    cout<<"CPU Elapsed time is: "<<gpu_elapsed_time<<" milliseconds"<<endl;
 
    cout<<"CPU Result : "<<endl;
    print_matrix(c, m, k);
    
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    
    delete[] a;
    delete[] b;
    delete[] c;
    
    return 0;
}